﻿#include "NestedMonteCarloVaR.h"

int main()
{
	const int path_ext = 32;  // Number of the outer MC loops
	const int path_int = 16;  // Number of the inner MC loops

	const int var_t = 1;					// VaR duration
	const float var_per = 0.95f;				// 1-percentile

	const int port_n = 2;					// Number of products in the portfolio
	float port_w[2] = { 0.5f, 0.5f };		// Weights of the products in the portfolio
											// { bond, stock }

	const float bond_par = 1000.0f;			// Par value of bond
	const float bond_c = 100.0f;			// Coupon
	const int bond_m = 10;					// Maturity
	const float bond_y = 0.08f;				// yeild
	const float bond_x = 1134.2f;			// today's price

	const float stock_s0 = 300.0f;			// Start value of stock
	const float stock_mu = 0.01f;			// risk free(or mean)
	const float stock_var = 0.13;			// Volatility
	const int stock_t = 1;					// Steps(trade days)
	const int stock_x = 20;					// Number of shares
    
		
	NestedMonteCarloVaR* mc = new NestedMonteCarloVaR(path_ext,path_int,var_t, var_per, port_n, port_w);
	mc->bond_init(bond_par, bond_c, bond_m, bond_y, bond_x, 0);
	mc->stock_init(stock_s0, stock_mu, stock_var, stock_t, stock_x, 1);
	//mc->execute();
    return 0;
}

/*****************************************************************************
 *                               fir_test.cpp
 *
 * FIR class testing.
 *
 * Zhang Ming, 2010-03
 *****************************************************************************/

//
//#define BOUNDS_CHECK
//
//#include <iostream>
//#include <fir.h>
//
//
//using namespace std;
//using namespace itlab;
