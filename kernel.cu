﻿#include "NestedMonteCarloVaR.h"

int main()
{
	const int path_ext = 5;  // Number of the outer MC loops
	const int path_int = 2;  // Number of the inner MC loops

	const int var_t = 1;					// VaR duration
	const float var_per = 0.95f;				// 1-percentile

	const int port_n = 3;					// Number of products in the portfolio
	float port_w[port_n] = { 0.0f, 0.0f, 1.0f };		// Weights of the products in the portfolio
											// { bond, stock }

	const float bond_par = 1000.0f;			// Par value of bond
	const float bond_c = 100.0f;			// Coupon
	const int bond_m = 10;					// Maturity
	const float bond_y = 0.08f;				// yeild
	const float bond_x = 1134.2f;			// today's price

	const float stock_s0 = 300.0f;			// Start value of stock
	const float stock_mu = 0.01f;			// risk free(or mean)
	const float stock_var = 0.13f;			// Volatility
	const int stock_t = 1;					// Steps(trade days)
	const int stock_x = 20;					// Number of shares

	Stock* s1 = new Stock(stock_s0, stock_mu, stock_var, stock_t, 100);
	Stock* s2 = new Stock(stock_s0, stock_mu, stock_var, stock_t, 100);
	const int bskop_n = 2;
	const float bskop_k = 31000.0f;
	Stock bskop_stocks[bskop_n] = { *s1, *s2 };
	float bskop_cov[bskop_n * bskop_n] = { 1.0f, 0.5f,
										   0.5f, 1.0f };
	float bskop_w[bskop_n] = { 0.5f, 0.5f };
		
	NestedMonteCarloVaR* mc = new NestedMonteCarloVaR(path_ext,path_int,var_t, var_per, port_n, port_w);
	mc->bond_init(bond_par, bond_c, bond_m, bond_y, bond_x, 0);
	mc->stock_init(stock_s0, stock_mu, stock_var, stock_t, stock_x, 1);
	mc->bskop_init(bskop_n, bskop_stocks, bskop_cov, bskop_k, bskop_w, 2);
	mc->execute();
    return 0;
}