﻿#include "NestedMonteCarloVaR.h"

int main()
{
	const int path_ext = 32;  // Number of the outer MC loops
	const int path_int = 16;  // Number of the inner MC loops

	const int var_t = 1;					// VaR duration
	const float var_per = 0.95f;				// 1-percentile

	const int port_n = 1;					// Number of products in the portfolio
	float port_w[1] = { 1.0 };				// Weights of the products in the portfolio
											// { bond }

	const float bond_par = 1000;			// Par value of bond
	const float bond_c = 100;				// Coupon
	const int bond_m = 10;					// Maturity
	const float bond_y = 0.08f;				// yeild
	const float bond_x = 1134.2f;			// today's price


    NestedMonteCarloVaR* mc = new NestedMonteCarloVaR(path_ext,path_int,var_t, var_per, port_n, port_w);
	mc->bond_init(bond_par, bond_c, bond_m, bond_y, bond_x, 0);
	mc->execute();
    return 0;
}
