﻿#include "NestedMonteCarloVaR.h"

int main()
{
	const int path_ext = 5;  // Number of the outer MC loops
	const int path_int = 2;  // Number of the inner MC loops

	const int var_t = 1;					// VaR duration
	const float var_per = 0.95f;				// 1-percentile

	const int port_n = 3;					// Number of products in the portfolio
	float port_w[port_n] = { 1.0f, 0.0f, 0.0f };		// Weights of the products in the portfolio
														// { bond, stock, basket option}
	const float risk_free = 0.02f;

	const float bond_par = 1000.0f;			// Par value of bond
	const float bond_c = 100.0f;			// Coupon
	const int bond_m = 10;					// Maturity
	float bond_y[bond_m] = { 
			5.00f, 5.69f, 6.09f, 6.38f, 6.61f,
			6.79f, 6.94f, 7.07f, 7.19f, 7.30f 
	};										// yeild curve
	const float sigma = 1.5f;				// sigma

	const float stock_s0 = 300.0f;			// Start value of stock
	const float stock_mu = risk_free;			// risk free(or mean)
	const float stock_var = 0.13f;			// Volatility
	const int stock_t = 1;					// Steps(trade days)
	const int stock_x = 20;					// Number of shares

	Stock* s1 = new Stock(stock_s0, stock_mu, stock_var, stock_t, 100);
	Stock* s2 = new Stock(stock_s0, stock_mu, stock_var, stock_t, 100);
	const int bskop_n = 2;
	const float bskop_k = 31000.0f;
	Stock bskop_stocks[bskop_n] = { *s1, *s2 };
	float bskop_cov[bskop_n * bskop_n] = { 1.0f, 0.5f,
										   0.5f, 1.0f };
	float bskop_w[bskop_n] = { 0.5f, 0.5f };
		
	NestedMonteCarloVaR* mc = new NestedMonteCarloVaR(
		path_ext,path_int,
		var_t, var_per,
		port_n, port_w,
		risk_free
	);
	mc->bond_init(bond_par, bond_c, bond_m, bond_y, sigma, 0);
	mc->stock_init(stock_s0, stock_mu, stock_var, stock_t, stock_x, 1);
	mc->bskop_init(bskop_n, bskop_stocks, bskop_cov, bskop_k, bskop_w, 2);
	mc->execute();
    return 0;
}